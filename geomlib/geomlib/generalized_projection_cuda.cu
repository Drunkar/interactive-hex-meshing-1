#include "hip/hip_runtime.h"
// clang-format off
#include "generalized_projection.h"
// clang-format on

#include <ATen/Functions.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "common.cuh"
#include "utils.cuh"
#include "vec_utils.cuh"

namespace geomlib {
    namespace {
        const float kEps = 1e-8;
        template <int dim, typename scalar_t>
            __device__ inline void ComputeBarycentricGradient(
                    const scalar_t* e1, const scalar_t* e2, const scalar_t e1_dot_e2,
                    const scalar_t e1_norm_sqr, const scalar_t e2_norm_sqr, scalar_t* grad_w1,
                    scalar_t* grad_w2) {
                {
                    zero_out_vec<dim>(grad_w1);
                    add_vec<dim>(grad_w1, e1);
                    scalar_t tmp[dim];
                    scalar_times_vec<dim>(-e1_dot_e2 / (kEps + e2_norm_sqr), e2, tmp);
                    add_vec<dim>(grad_w1, tmp);
                    scalar_t denom = e1_norm_sqr - e1_dot_e2 * e1_dot_e2 / (kEps + e2_norm_sqr);
                    scalar_times_vec<dim>(1 / (denom + kEps), grad_w1, grad_w1);
                }
                {
                    zero_out_vec<dim>(grad_w2);
                    add_vec<dim>(grad_w2, e2);
                    scalar_t tmp[dim];
                    scalar_times_vec<dim>(-e1_dot_e2 / (kEps + e1_norm_sqr), e1, tmp);
                    add_vec<dim>(grad_w2, tmp);
                    scalar_t denom = e2_norm_sqr - e1_dot_e2 * e1_dot_e2 / (kEps + e1_norm_sqr);
                    scalar_times_vec<dim>(1 / (denom + kEps), grad_w2, grad_w2);
                }
            }

        template <int dim, typename scalar_t>
            __device__ void GeneralizedTriangleProjection(
                    const scalar_t* p, const scalar_t* v0, const scalar_t* e1,
                    const scalar_t* e2, const scalar_t e1_dot_e2, const scalar_t e1_norm_sqr,
                    const scalar_t e2_norm_sqr, const scalar_t* grad_w1,
                    const scalar_t* grad_w2,
                    // Below are results:
                    scalar_t* result_dist, scalar_t* result_w1, scalar_t* result_w2) {
                scalar_t w1, w2;
                scalar_t
                    p_minus_v0[dim];  // this array is unavoidable since we need to compute
                // this for every pair of query point and face

                bool degenerate = false;
                minus_vec<dim>(p, v0, p_minus_v0);
                if (grad_w1 == nullptr || grad_w2 == nullptr) {
                    scalar_t b1 = dot_vec<dim>(e1, p_minus_v0);
                    scalar_t b2 = dot_vec<dim>(e2, p_minus_v0);

                    scalar_t det = e1_norm_sqr * e2_norm_sqr - e1_dot_e2 * e1_dot_e2;
                    if (det < -kEps || det > kEps) {
                        // Cramer's rule.
                        w1 = (b1 * e2_norm_sqr - b2 * e1_dot_e2) / det;
                        w2 = (b2 * e1_norm_sqr - b1 * e1_dot_e2) / det;
                    } else {
                        degenerate = true;
                    }
                } else {
                    w1 = dot_vec<dim>(grad_w1, p_minus_v0);
                    w2 = dot_vec<dim>(grad_w2, p_minus_v0);
                }


                if (!degenerate && 0 <= 1 - w1 - w2 && 0 <= w1 && 0 <= w2) {
                    scalar_t p_proj[dim];
                    {
                        {
                            scalar_t w1_e1[dim];
                            scalar_times_vec<dim>(w1, e1, w1_e1);
                            scalar_times_vec<dim>(w2, e2, p_proj);
                            add_vec<dim>(p_proj, w1_e1);
                        }
                    }
                    *result_dist = distance_sqr_vec<dim>(p_minus_v0, p_proj);
                    *result_w1 = w1;
                    *result_w2 = w2;
                } else {
                    // Project to three edges.
                    scalar_t w1_tmp[3];
                    scalar_t w2_tmp[3];

                    w1_tmp[0] = clamp01(dot_vec<dim>(p_minus_v0, e1) / (kEps + e1_norm_sqr));
                    w2_tmp[0] = 0;

                    w2_tmp[1] = clamp01(dot_vec<dim>(p_minus_v0, e2) / (kEps + e2_norm_sqr));
                    w1_tmp[1] = 0;

                    {
                        scalar_t numer = dot_vec<dim>(p_minus_v0, e2) -
                            dot_vec<dim>(p_minus_v0, e1) - e1_dot_e2 + e1_norm_sqr;
                        scalar_t denom = e1_norm_sqr + e2_norm_sqr - 2 * e1_dot_e2;
                        w2_tmp[2] = clamp01(numer / (kEps + denom));
                        w1_tmp[2] = 1 - w2_tmp[2];
                    }

                    scalar_t best_dist = FLT_MAX;
                    int best_k = -1;

                    for (int k = 0; k < 3; k++) {
                        scalar_t p_proj[dim];
                        scalar_t w1_e1[dim];
                        scalar_times_vec<dim>(w1_tmp[k], e1, w1_e1);
                        scalar_times_vec<dim>(w2_tmp[k], e2, p_proj);
                        add_vec<dim>(p_proj, w1_e1);
                        scalar_t dist = distance_sqr_vec<dim>(p_minus_v0, p_proj);
                        if (dist < best_dist) {
                            best_dist = dist;
                            best_k = k;
                        }
                    }

                    *result_dist = best_dist;
                    *result_w1 = w1_tmp[best_k];
                    *result_w2 = w2_tmp[best_k];
                }
            }


        template <int dim, typename scalar_t>
            __device__ void GeneralizedTetrahedronProjection(
                    const scalar_t* p,              // D
                    const scalar_t* v0,             // D
                    const scalar_t* e_mat,          // 3xD,
                    const scalar_t* e_dot_mat,      // 3x3
                    const scalar_t* e_dot_inv_mat,  // 3x3
                    const scalar_t is_degenerate,   // float, 1.0 if degenerate
                    // Below are results:
                    scalar_t* result_dist,    // scalar
                    scalar_t* result_weights  // 3
                    ) {
                bool recurse = true;
                {
                    scalar_t p_minus_v0[dim];
                    minus_vec<dim>(p, v0, p_minus_v0);
                    scalar_t b[3];
                    for (int i = 0; i < 3; i++) {
                        b[i] = dot_vec<dim>(&e_mat[i * dim], p_minus_v0);
                    }
                    if (is_degenerate < 0.5) {
                        scalar_t weights[3];
                        mat_vec_mult<3>(e_dot_inv_mat, b, weights);
                        scalar_t weight_op = 1 - weights[0] - weights[1] - weights[2];
                        if (weights[0] >= 0 && weights[1] >= 0 && weights[2] >= 0
                                && weight_op >= 0) {
                            recurse = false;
                            copy_vec<3>(result_weights, weights);

                            scalar_t p_proj[dim];
                            zero_out_vec<dim>(p_proj);
                            for (int i = 0; i < 3; i++) {
                                scalar_t tmp[dim];
                                scalar_times_vec<dim>(result_weights[i], &e_mat[i * dim], tmp);
                                add_vec<dim>(p_proj, tmp);
                            }
                            *result_dist = distance_sqr_vec<dim>(p, p_proj);
                        }
                    }
                }
                if (recurse) {
                    // Calculate weights by projecting onto each of 4 faces.
                    *result_dist = FLT_MAX;
                    for (int i = 0; i < 4; i++) {
                        scalar_t vc[dim];
                        scalar_t ecj[dim];
                        scalar_t eck[dim];
                        scalar_t ecj_dot_eck;
                        scalar_t ecj_norm_sqr;
                        scalar_t eck_norm_sqr;
                        if (i < 3) {
                            copy_vec<dim>(vc, v0);
                            int j = i;
                            int k = (i + 1) % 3;
                            copy_vec<dim>(ecj, &e_mat[j * dim]);
                            copy_vec<dim>(eck, &e_mat[k * dim]);
                            ecj_dot_eck = e_dot_mat[j * 3 + k];
                            ecj_norm_sqr = e_dot_mat[j * 3 + j];
                            eck_norm_sqr = e_dot_mat[k * 3 + k];
                        } else {
                            plus_vec<dim>(v0, &e_mat[2 * dim], vc);
                            minus_vec<dim>(&e_mat[0 * dim], &e_mat[2 * dim], ecj);
                            minus_vec<dim>(&e_mat[1 * dim], &e_mat[2 * dim], eck);
                            ecj_dot_eck = dot_vec<dim>(ecj, eck);
                            ecj_norm_sqr = dot_vec<dim>(ecj, ecj);
                            eck_norm_sqr = dot_vec<dim>(eck, eck);
                        }
                        scalar_t cur_dist;
                        scalar_t cur_weights[2];
                        GeneralizedTriangleProjection<dim>(
                                p, vc, ecj, eck, ecj_dot_eck, ecj_norm_sqr, eck_norm_sqr,
                                (const scalar_t*)nullptr, (const scalar_t*)nullptr,
                                &cur_dist, &cur_weights[0], &cur_weights[1]);
                        if (cur_dist < *result_dist) {
                            *result_dist = cur_dist;
                            zero_out_vec<3>(result_weights);
                            if (i < 3) {
                                result_weights[i] = cur_weights[0];
                                result_weights[(i + 1) % 3] = cur_weights[1];
                            } else {
                                result_weights[0] = cur_weights[0];
                                result_weights[1] = cur_weights[1];
                                result_weights[2] = 1 - cur_weights[0] - cur_weights[1];
                            }
                        }
                    }
                }
            }

        template <int dim, typename scalar_t>
            __global__ void GeneralizedTriangleProjectionKernel(
                    const scalar_t* __restrict__ points,  // P x D
                    const size_t num_faces,
                    // Pre-computed values:
                    const scalar_t* __restrict__ v0,           // FxD, vertex 0 of faces
                    const scalar_t* __restrict__ e1,           // FxD, vertex 1 - vertex 0
                    const scalar_t* __restrict__ e2,           // FxD
                    const scalar_t* __restrict__ e1_dot_e2,    // F
                    const scalar_t* __restrict__ e1_norm_sqr,  // F
                    const scalar_t* __restrict__ e2_norm_sqr,  // F
                    const scalar_t* __restrict__ grad_w1,      // FxD, dw1/dp on the face
                    const scalar_t* __restrict__ grad_w2,      // FxD
                    // Results:
                    scalar_t* __restrict__ result_dists, int* __restrict__ result_idxs,
                    scalar_t* __restrict__ result_w1, scalar_t* __restrict__ result_w2) {
                extern __shared__ char shared_buf[];
                scalar_t* min_dists = (scalar_t*)shared_buf;         // scalar_t[num_threads]
                size_t* min_tids = (size_t*)&min_dists[blockDim.x];  // size_t[num_threads]

                // Each block computing the projection of a single point.
                const size_t i = blockIdx.x;
                // Faces are divided evenly among the threads.
                const size_t tid = threadIdx.x;

                const scalar_t* p = &points[i * dim];  // point to project

                scalar_t min_dist = FLT_MAX;
                size_t min_idx = 0;
                scalar_t min_w1;
                scalar_t min_w2;

                for (int j = tid; j < num_faces; j += blockDim.x) {
                    scalar_t dist, w1, w2;
                    GeneralizedTriangleProjection<dim, scalar_t>(
                            p, &v0[j * dim], &e1[j * dim], &e2[j * dim], e1_dot_e2[j],
                            e1_norm_sqr[j], e2_norm_sqr[j], &grad_w1[j * dim], &grad_w2[j * dim],
                            &dist, &w1, &w2);

                    if (dist < min_dist) {
                        min_dist = dist;
                        min_idx = j;
                        min_w1 = w1;
                        min_w2 = w2;
                    }
                }
                min_dists[tid] = min_dist;
                min_tids[tid] = tid;
                __syncthreads();

                ReduceMin(min_dists, min_tids, tid);

                __syncthreads();

                // Finally thread with min_dist writes the result to the output.
                if (tid == min_tids[0]) {
                    result_dists[i] = min_dist;  // squared minimum distance
                    result_idxs[i] = min_idx;
                    assert(min_idx < num_faces);
                    result_w1[i] = min_w1;
                    result_w2[i] = min_w2;
                }
            }

        template <int dim, typename scalar_t>
            __global__ void GeneralizedTetrahedronProjectionKernel(
                    const scalar_t* __restrict__ points,  // P x D
                    const size_t num_tets,
                    // Pre-computed values:
                    const scalar_t* __restrict__ v0,             // TxD, vertex 0 of faces
                    const scalar_t* __restrict__ e_mat,          // Tx3xD, vertex i - vertex 0
                    const scalar_t* __restrict__ e_dot_mat,      // Tx3x3
                    const scalar_t* __restrict__ e_dot_inv_mat,  // Tx3x3
                    const scalar_t* __restrict__ is_degenerate,  // T
                    // Results:
                    scalar_t* __restrict__ result_dists, int* __restrict__ result_idxs,
                    scalar_t* __restrict__ result_weights  // Px3
                    ) {
                extern __shared__ char shared_buf[];
                scalar_t* min_dists = (scalar_t*)shared_buf;         // scalar_t[num_threads]
                size_t* min_tids = (size_t*)&min_dists[blockDim.x];  // size_t[num_threads]

                // Each block computing the projection of a single point.
                const size_t i = blockIdx.x;
                // Faces are divided evenly among the threads.
                const size_t tid = threadIdx.x;

                const scalar_t* p = &points[i * dim];  // point to project

                scalar_t min_dist = FLT_MAX;
                size_t min_idx = 0;
                scalar_t min_weights[3];

                for (int j = tid; j < num_tets; j += blockDim.x) {
                    scalar_t dist;
                    scalar_t weights[3];
                    GeneralizedTetrahedronProjection<dim, scalar_t>(
                            p, &v0[j * dim], &e_mat[j * 3 * dim], &e_dot_mat[j * 3 * 3],
                            &e_dot_inv_mat[j * 3 * 3], is_degenerate[j],
                            &dist, weights);

                    if (dist < min_dist) {
                        min_dist = dist;
                        min_idx = j;
                        copy_vec<3>(min_weights, weights);
                    }
                }
                min_dists[tid] = min_dist;
                min_tids[tid] = tid;
                __syncthreads();

                ReduceMin(min_dists, min_tids, tid);

                // Finally thread with min_dist writes the result to the output.
                if (tid == min_tids[0]) {
                    result_dists[i] = min_dist;  // squared minimum distance
                    result_idxs[i] = min_idx;
                    assert(min_idx < num_tets);
                    copy_vec<3>(&result_weights[i * 3], min_weights);
                }
            }
    }  // namespace

    template <int dim>
        std::vector<torch::Tensor> ComputeGeneralizedTriangleProjection(
                torch::Tensor points, const TriangularProjectionInfo& info) {
            CHECK_INPUT(points);

            TORCH_CHECK(points.size(1) == dim);
            TORCH_CHECK(info.dim == dim);
            TORCH_CHECK(points.dtype() == torch::kFloat32 ||
                    points.dtype() == torch::kFloat64);
            TORCH_CHECK(points.dtype() == info.v0.dtype());

            at::cuda::CUDAGuard device_guard{points.device()};
            hipStream_t stream = at::cuda::getCurrentCUDAStream();

            int num_points = points.size(0);
            int num_faces = info.num_faces;

            torch::Tensor result_dists =
                torch::zeros({num_points}, points.options()).contiguous();
            torch::Tensor result_idxs =
                torch::zeros({num_points}, points.options().dtype(torch::kInt32))
                .contiguous();
            torch::Tensor result_w1 =
                torch::zeros({num_points}, points.options()).contiguous();
            torch::Tensor result_w2 =
                torch::zeros({num_points}, points.options()).contiguous();

            size_t num_threads = 128;
            dim3 num_blocks(num_points);
            size_t shared_size =
                num_threads *
                ((points.dtype() == torch::kFloat32 ? sizeof(float) : sizeof(double)) +
                 sizeof(size_t));

            AT_DISPATCH_FLOATING_TYPES(
                    points.scalar_type(), "ComputeGeneralizedTriangleProjection", [&] {
                    GeneralizedTriangleProjectionKernel<dim, scalar_t>
                    <<<num_blocks, num_threads, shared_size, stream>>>(
                            points.contiguous().data_ptr<scalar_t>(),
                            static_cast<size_t>(num_faces),
                            info.v0.contiguous().data_ptr<scalar_t>(),
                            info.e1.contiguous().data_ptr<scalar_t>(),
                            info.e2.contiguous().data_ptr<scalar_t>(),
                            info.e1_dot_e2.contiguous().data_ptr<scalar_t>(),
                            info.e1_norm_sqr.contiguous().data_ptr<scalar_t>(),
                            info.e2_norm_sqr.contiguous().data_ptr<scalar_t>(),
                            info.grad_w1.contiguous().data_ptr<scalar_t>(),
                            info.grad_w2.contiguous().data_ptr<scalar_t>(),
                            result_dists.data_ptr<scalar_t>(), result_idxs.data_ptr<int>(),
                            result_w1.data_ptr<scalar_t>(), result_w2.data_ptr<scalar_t>());
                    });

            AT_CUDA_CHECK(hipGetLastError());

            return {result_dists, result_idxs.to(torch::kInt64), result_w1, result_w2};
        }

    template <int dim>
        std::vector<torch::Tensor> ComputeGeneralizedTetrahedronProjection(
                torch::Tensor points, torch::Tensor vertices, torch::Tensor tets) {
            CHECK_INPUT(points);
            CHECK_INPUT(vertices);
            CHECK_INPUT(tets);

            TORCH_CHECK(points.size(1) == dim);
            TORCH_CHECK(vertices.size(1) == dim);
            TORCH_CHECK(points.dtype() == torch::kFloat32 ||
                    points.dtype() == torch::kFloat64);
            TORCH_CHECK(tets.dtype() ==
                    torch::kInt64);  // torch requires 64-bit int for indexing
            TORCH_CHECK(points.dtype() == vertices.dtype());

            at::cuda::CUDAGuard device_guard{points.device()};
            hipStream_t stream = at::cuda::getCurrentCUDAStream();

            // Pre-compute reusable values.
            using namespace torch::indexing;
            std::vector<torch::Tensor> v_list;
            for (int i = 0; i < 4; i++) {
                v_list.push_back(vertices.index({tets.index({Slice(), i}), Slice()}));
            }
            std::vector<torch::Tensor> e_list;
            for (int i = 1; i < 4; i++) {
                e_list.push_back(v_list[i] - v_list[0]);
            }
            std::vector<torch::Tensor> e_dot_list;
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    e_dot_list.push_back((e_list[i] * e_list[j]).sum(-1));
                }
            }
            auto e_mat = torch::stack(e_list, 1);                              // Tx3xD
            auto e_dot_mat = torch::stack(e_dot_list, 1).reshape({-1, 3, 3});  // Tx3x3

            auto e_dot_mat_det = torch::linalg_det(e_dot_mat); // T
            auto is_degenerate = (e_dot_mat_det.abs() < kEps).to(points.dtype()); // T
            auto e_dot_inv_mat = torch::linalg_pinv(e_dot_mat);  // Tx3x3

            size_t num_points = points.size(0);
            size_t num_tets = tets.size(0);

            torch::Tensor result_dists =
                torch::zeros({static_cast<int>(num_points)}, points.options())
                .contiguous();
            torch::Tensor result_idxs = torch::zeros({static_cast<int>(num_points)},
                    tets.options().dtype(torch::kInt32))
                .contiguous();
            torch::Tensor result_weights =
                torch::zeros({static_cast<int>(num_points), 3},
                        points.options())
                .contiguous();  // w1, w2, w3 of barycentric coordinates

            size_t num_threads = 128;
            dim3 num_blocks(num_points);
            size_t shared_size =
                num_threads *
                ((points.dtype() == torch::kFloat32 ? sizeof(float) : sizeof(double)) +
                 sizeof(size_t));

            AT_DISPATCH_FLOATING_TYPES(
                    points.scalar_type(), "ComputeGeneralizedTetrahedronProjection", [&] {
                    GeneralizedTetrahedronProjectionKernel<dim, scalar_t>
                    <<<num_blocks, num_threads, shared_size, stream>>>(
                            points.contiguous().data_ptr<scalar_t>(), num_tets,
                            v_list[0].contiguous().data_ptr<scalar_t>(),
                            e_mat.contiguous().data_ptr<scalar_t>(),
                            e_dot_mat.contiguous().data_ptr<scalar_t>(),
                            e_dot_inv_mat.contiguous().data_ptr<scalar_t>(),
                            is_degenerate.contiguous().data_ptr<scalar_t>(),
                            result_dists.data_ptr<scalar_t>(), result_idxs.data_ptr<int>(),
                            result_weights.data_ptr<scalar_t>());
                    });

            AT_CUDA_CHECK(hipGetLastError());

            return {result_dists, result_idxs.to(torch::kInt64), result_weights};
        }

    // Explicit instantiations.
    template std::vector<torch::Tensor> ComputeGeneralizedTriangleProjection<3>(
            torch::Tensor points, const TriangularProjectionInfo& info);
    template std::vector<torch::Tensor> ComputeGeneralizedTriangleProjection<8>(
            torch::Tensor points, const TriangularProjectionInfo& info);
    template std::vector<torch::Tensor> ComputeGeneralizedTetrahedronProjection<3>(
            torch::Tensor points, torch::Tensor vertices, torch::Tensor tets);

}  // namespace geomlib
